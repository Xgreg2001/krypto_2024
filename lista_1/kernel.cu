#include <cstddef>
#include <cstdint>
#include <cstdio>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <sys/types.h>

#define STATE_SIZE_WORDS 4
#define CANDIDATES_SIZE_WORDS 16

#define A1_ONE_BITS 0x84200000u
#define A1_ZERO_BITS 0x0A000820u
#define D1_ONE_BITS 0x8C000800u
#define D1_ZERO_BITS 0x02208026u
#define D1_A1_SAME_BITS 0x701F10C0u
#define C1_ONE_BITS 0xBE1F0966u
#define C1_ZERO_BITS 0x40201080u
#define C1_D1_SAME_BITS 0x00000018u
#define B1_ONE_BITS 0xBA040010u
#define B1_ZERO_BITS 0x443B19EEu
#define B1_C1_SAME_BITS 0x00000601u
#define A2_ONE_BITS 0x482F0E50u
#define A2_ZERO_BITS 0xB41011AFu
#define D2_ONE_BITS 0x04220C56u
#define D2_ZERO_BITS 0x9A1113A9u
#define C2_ONE_BITS 0x96011E01u
#define C2_ZERO_BITS 0x083201C0u
#define C2_D2_SAME_BITS 0x01808000u
#define B2_ONE_BITS 0x843283C0u
#define B2_ZERO_BITS 0x1B810001u
#define B2_C2_SAME_BITS 0x00000002u
#define A3_ONE_BITS 0x9C0101C1u
#define A3_ZERO_BITS 0x03828202u
#define A3_B2_SAME_BITS 0x00001000u
#define D3_ONE_BITS 0x878383C0u
#define D3_ZERO_BITS 0x00041003u
#define C3_ONE_BITS 0x800583C3u
#define C3_ZERO_BITS 0x00021000u
#define C3_D3_SAME_BITS 0x00086000u
#define B3_ONE_BITS 0x80081080u
#define B3_ZERO_BITS 0x0007E000u
#define B3_C3_SAME_BITS 0x7F000000u
#define A4_ONE_BITS 0x3F0FE008u
#define A4_ZERO_BITS 0xC0000080u
#define D4_ONE_BITS 0x400BE088u
#define D4_ZERO_BITS 0xBF040000u
#define C4_ONE_BITS 0x7D000000u
#define C4_ZERO_BITS 0x82008008u
#define B4_ONE_BITS 0x20000000u
#define B4_ZERO_BITS 0x80000000u
#define A5_ZERO_BITS 0x80020000u
#define A5_B4_SAME_BITS 0x00008008u
#define D5_ONE_BITS 0x00020000u
#define D5_ZERO_BITS 0x80000000u
#define D5_A5_SAME_BITS 0x20000000u
#define C5_ZERO_BITS 0x80020000u
#define B5_ZERO_BITS 0x80000000u
#define A6_ZERO_BITS 0x80000000u
#define A6_B5_SAME_BITS 0x00020000u
#define D6_ZERO_BITS 0x80000000u
#define C6_ZERO_BITS 0x80000000u
#define B6_C6_DIFFERENT_BITS 0x80000000u
#define B12_D12_SAME_BITS 0x80000000u
#define A13_C12_SAME_BITS 0x80000000u
#define D13_B12_DIFFERENT_BITS 0x80000000u
#define C13_A13_SAME_BITS 0x80000000u
#define B13_D13_SAME_BITS 0x80000000u
#define A14_C13_SAME_BITS 0x80000000u
#define D14_B13_SAME_BITS 0x80000000u
#define C14_A14_SAME_BITS 0x80000000u
#define B14_D14_SAME_BITS 0x80000000u
#define A15_C14_SAME_BITS 0x80000000u
#define D15_B14_SAME_BITS 0x80000000u
#define C15_A15_SAME_BITS 0x80000000u
#define B15_D15_DIFFERENT_BITS 0x80000000u
#define A16_ONE_BITS 0x02000000u
#define A16_C15_SAME_BITS 0x80000000u
#define D16_ONE_BITS 0x02000000u
#define D16_B15_SAME_BITS 0x80000000u

__device__ uint32_t _rotl(const uint32_t value, uint32_t shift) {
  if ((shift &= sizeof(value) * 8 - 1) == 0)
    return value;
  return (value << shift) | (value >> (sizeof(value) * 8 - shift));
}

__device__ uint32_t _rotr(const uint32_t value, uint32_t shift) {
  if ((shift &= sizeof(value) * 8 - 1) == 0)
    return value;
  return (value >> shift) | (value << (sizeof(value) * 8 - shift));
}

__device__ uint32_t F(uint32_t x, uint32_t y, uint32_t z) {
  return (x & y) | ((~x) & z);
}

__device__ uint32_t G(uint32_t x, uint32_t y, uint32_t z) {
  return (x & z) | (y & ~z);
}

__device__ uint32_t H(uint32_t x, uint32_t y, uint32_t z) { return x ^ y ^ z; }

__device__ uint32_t I(uint32_t x, uint32_t y, uint32_t z) {
  return y ^ (x | ~z);
}

__device__ uint32_t FF(uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                       uint32_t word, uint32_t k, uint32_t s) {
  uint32_t f = a + F(b, c, d) + word + k;
  return _rotl(f, s) + b;
}

__device__ uint32_t GG(uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                       uint32_t word, uint32_t k, uint32_t s) {
  uint32_t g = a + G(b, c, d) + word + k;
  return _rotl(g, s) + b;
}

__device__ uint32_t HH(uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                       uint32_t word, uint32_t k, uint32_t s) {
  uint32_t h = a + H(b, c, d) + word + k;
  return _rotl(h, s) + b;
}

__device__ uint32_t II(uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                       uint32_t word, uint32_t k, uint32_t s) {
  uint32_t i = a + I(b, c, d) + word + k;
  return _rotl(i, s) + b;
}

__device__ uint32_t apply_one_bits(uint32_t v, uint32_t mask) {
  return v | mask;
}

__device__ uint32_t apply_zero_bits(uint32_t v, uint32_t mask) {
  return v & (~mask);
}

__device__ uint32_t apply_same_bits(uint32_t v, uint32_t u, uint32_t mask) {
  return (v | (u & mask)) & (u | (~mask));
}

__device__ bool verify_one_bits(uint32_t v, uint32_t mask) {
  return (v & mask) == mask;
}

__device__ bool verify_zero_bits(uint32_t v, uint32_t mask) {
  return (v & mask) == 0;
}

__device__ bool verify_same_bits(uint32_t v, uint32_t u, uint32_t mask) {
  return (v & mask) == (u & mask);
}

__device__ bool verify_different_bits(uint32_t v, uint32_t u, uint32_t mask) {
  return (v & mask) != (u & mask);
}

__device__ uint32_t reverse_FF(uint32_t a, uint32_t b, uint32_t c, uint32_t d,
                               uint32_t t, uint32_t s, uint32_t orig) {
  return _rotr((a - b), s) - F(b, c, d) - orig - t;
}

__global__ void generate_candidates(const uint32_t md5_state[STATE_SIZE_WORDS],
                                    uint32_t *candidates, bool *found,
                                    size_t iterations, size_t seed) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  hiprandState random_state;
  hiprand_init(seed, idx, 0, &random_state);

  uint32_t words[16];

  for (size_t i = 0; i < iterations; i++) {

    for (int j = 0; j < 16; ++j) {
      words[j] = hiprand(&random_state);
    }

    uint32_t a = md5_state[0];
    uint32_t b = md5_state[1];
    uint32_t c = md5_state[2];
    uint32_t d = md5_state[3];

    uint32_t orig;

    // ROUND 1

    //   a1
    orig = a;
    a = FF(a, b, c, d, words[0], 0xD76AA478u, 7u);
    a = apply_one_bits(a, A1_ONE_BITS);
    a = apply_zero_bits(a, A1_ZERO_BITS);
    words[0] = reverse_FF(a, b, c, d, 0xD76AA478, 7, orig);

    // d1
    orig = d;
    d = FF(d, a, b, c, words[1], 0xE8C7B756, 12);
    d = apply_one_bits(d, D1_ONE_BITS);
    d = apply_zero_bits(d, D1_ZERO_BITS);
    d = apply_same_bits(d, a, D1_A1_SAME_BITS);
    words[1] = reverse_FF(d, a, b, c, 0xE8C7B756, 12, orig);

    // c1
    orig = c;
    c = FF(c, d, a, b, words[2], 0x242070DB, 17);
    c = apply_one_bits(c, C1_ONE_BITS);
    c = apply_zero_bits(c, C1_ZERO_BITS);
    c = apply_same_bits(c, d, C1_D1_SAME_BITS);
    words[2] = reverse_FF(c, d, a, b, 0x242070DB, 17, orig);

    // b1
    orig = b;
    b = FF(b, c, d, a, words[3], 0xC1BDCEEE, 22);
    b = apply_one_bits(b, B1_ONE_BITS);
    b = apply_zero_bits(b, B1_ZERO_BITS);
    b = apply_same_bits(b, c, B1_C1_SAME_BITS);
    words[3] = reverse_FF(b, c, d, a, 0xC1BDCEEE, 22, orig);

    // a2
    orig = a;
    a = FF(a, b, c, d, words[4], 0xF57C0FAF, 7);
    a = apply_one_bits(a, A2_ONE_BITS);
    a = apply_zero_bits(a, A2_ZERO_BITS);
    words[4] = reverse_FF(a, b, c, d, 0xF57C0FAF, 7, orig);

    // d2
    orig = d;
    d = FF(d, a, b, c, words[5], 0x4787C62A, 12);
    d = apply_one_bits(d, D2_ONE_BITS);
    d = apply_zero_bits(d, D2_ZERO_BITS);
    words[5] = reverse_FF(d, a, b, c, 0x4787C62A, 12, orig);

    // c2
    orig = c;
    c = FF(c, d, a, b, words[6], 0xA8304613, 17);
    c = apply_one_bits(c, C2_ONE_BITS);
    c = apply_zero_bits(c, C2_ZERO_BITS);
    c = apply_same_bits(c, d, C2_D2_SAME_BITS);
    words[6] = reverse_FF(c, d, a, b, 0xA8304613, 17, orig);

    // b2
    orig = b;
    b = FF(b, c, d, a, words[7], 0xFD469501, 22);
    b = apply_one_bits(b, B2_ONE_BITS);
    b = apply_zero_bits(b, B2_ZERO_BITS);
    b = apply_same_bits(b, c, B2_C2_SAME_BITS);
    words[7] = reverse_FF(b, c, d, a, 0xFD469501, 22, orig);

    // a3
    orig = a;
    a = FF(a, b, c, d, words[8], 0x698098D8, 7);
    a = apply_one_bits(a, A3_ONE_BITS);
    a = apply_zero_bits(a, A3_ZERO_BITS);
    a = apply_same_bits(a, b, A3_B2_SAME_BITS);
    words[8] = reverse_FF(a, b, c, d, 0x698098D8, 7, orig);

    // d3
    orig = d;
    d = FF(d, a, b, c, words[9], 0x8B44F7AF, 12);
    d = apply_one_bits(d, D3_ONE_BITS);
    d = apply_zero_bits(d, D3_ZERO_BITS);
    words[9] = reverse_FF(d, a, b, c, 0x8B44F7AF, 12, orig);

    // c3
    orig = c;
    c = FF(c, d, a, b, words[10], 0xFFFF5BB1, 17);
    c = apply_one_bits(c, C3_ONE_BITS);
    c = apply_zero_bits(c, C3_ZERO_BITS);
    c = apply_same_bits(c, d, C3_D3_SAME_BITS);
    words[10] = reverse_FF(c, d, a, b, 0xFFFF5BB1, 17, orig);

    // b3
    orig = b;
    b = FF(b, c, d, a, words[11], 0x895CD7BE, 22);
    b = apply_one_bits(b, B3_ONE_BITS);
    b = apply_zero_bits(b, B3_ZERO_BITS);
    b = apply_same_bits(b, c, B3_C3_SAME_BITS);
    words[11] = reverse_FF(b, c, d, a, 0x895CD7BE, 22, orig);

    // a4
    orig = a;
    a = FF(a, b, c, d, words[12], 0x6B901122, 7);
    a = apply_one_bits(a, A4_ONE_BITS);
    a = apply_zero_bits(a, A4_ZERO_BITS);
    words[12] = reverse_FF(a, b, c, d, 0x6B901122, 7, orig);

    // d4
    orig = d;
    d = FF(d, a, b, c, words[13], 0xFD987193, 12);
    d = apply_one_bits(d, D4_ONE_BITS);
    d = apply_zero_bits(d, D4_ZERO_BITS);
    words[13] = reverse_FF(d, a, b, c, 0xFD987193, 12, orig);

    // c4
    orig = c;
    c = FF(c, d, a, b, words[14], 0xA679438E, 17);
    c = apply_one_bits(c, C4_ONE_BITS);
    c = apply_zero_bits(c, C4_ZERO_BITS);
    words[14] = reverse_FF(c, d, a, b, 0xA679438E, 17, orig);

    // b4
    orig = b;
    b = FF(b, c, d, a, words[15], 0x49B40821, 22);
    b = apply_one_bits(b, B4_ONE_BITS);
    b = apply_zero_bits(b, B4_ZERO_BITS);
    words[15] = reverse_FF(b, c, d, a, 0x49B40821, 22, orig);

    // ROUND 2

    // a5
    a = GG(a, b, c, d, words[1], 0xF61E2562, 5);
    if (!verify_zero_bits(a, A5_ZERO_BITS)) {
      continue;
    }
    if (!verify_same_bits(a, b, A5_B4_SAME_BITS)) {
      continue;
    }

    // d5
    d = GG(d, a, b, c, words[6], 0xC040B340, 9);
    if (!verify_zero_bits(d, D5_ZERO_BITS)) {
      continue;
    }
    if (!verify_one_bits(d, D5_ONE_BITS)) {
      continue;
    }
    if (!verify_same_bits(d, a, D5_A5_SAME_BITS)) {
      continue;
    }

    // c5
    c = GG(c, d, a, b, words[11], 0x265E5A51, 14);
    if (!verify_zero_bits(c, C5_ZERO_BITS)) {
      continue;
    }

    // b5
    b = GG(b, c, d, a, words[0], 0xE9B6C7AA, 20);
    if (!verify_zero_bits(b, B5_ZERO_BITS)) {
      continue;
    }

    // a6
    a = GG(a, b, c, d, words[5], 0xD62F105D, 5);
    if (!verify_zero_bits(a, A6_ZERO_BITS)) {
      continue;
    }
    if (!verify_same_bits(a, b, A6_B5_SAME_BITS)) {
      continue;
    }

    // d6
    d = GG(d, a, b, c, words[10], 0x02441453, 9);
    if (!verify_zero_bits(d, D6_ZERO_BITS)) {
      continue;
    }

    // c6
    c = GG(c, d, a, b, words[15], 0xD8A1E681, 14);
    if (!verify_zero_bits(c, C6_ZERO_BITS)) {
      continue;
    }

    // b6
    b = GG(b, c, d, a, words[4], 0xE7D3FBC8, 20);
    if (!verify_different_bits(b, c, B6_C6_DIFFERENT_BITS)) {
      continue;
    }

    a = GG(a, b, c, d, words[9], 0x21E1CDE6, 5);
    d = GG(d, a, b, c, words[14], 0xC33707D6, 9);
    c = GG(c, d, a, b, words[3], 0xF4D50D87, 14);
    b = GG(b, c, d, a, words[8], 0x455A14ED, 20);

    a = GG(a, b, c, d, words[13], 0xA9E3E905, 5);
    d = GG(d, a, b, c, words[2], 0xFCEFA3F8, 9);
    c = GG(c, d, a, b, words[7], 0x676F02D9, 14);
    b = GG(b, c, d, a, words[12], 0x8D2A4C8A, 20);

    // ROUND 3

    a = HH(a, b, c, d, words[5], 0xFFFA3942, 4);
    d = HH(d, a, b, c, words[8], 0x8771F681, 11);
    c = HH(c, d, a, b, words[11], 0x6D9D6122, 16);
    b = HH(b, c, d, a, words[14], 0xFDE5380C, 23);

    a = HH(a, b, c, d, words[1], 0xA4BEEA44, 4);
    d = HH(d, a, b, c, words[4], 0x4BDECFA9, 11);
    c = HH(c, d, a, b, words[7], 0xF6BB4B60, 16);
    b = HH(b, c, d, a, words[10], 0xBEBFBC70, 23);

    a = HH(a, b, c, d, words[13], 0x289B7EC6, 4);
    d = HH(d, a, b, c, words[0], 0xEAA127FA, 11);
    c = HH(c, d, a, b, words[3], 0xD4EF3085, 16);
    b = HH(b, c, d, a, words[6], 0x04881D05, 23);

    a = HH(a, b, c, d, words[9], 0xD9D4D039, 4);
    d = HH(d, a, b, c, words[12], 0xE6DB99E5, 11);
    c = HH(c, d, a, b, words[15], 0x1FA27CF8, 16);

    // b12
    b = HH(b, c, d, a, words[2], 0xC4AC5665, 23);
    if (!verify_same_bits(b, d, B12_D12_SAME_BITS)) {
      continue;
    }

    // ROUND 4

    // a13
    a = II(a, b, c, d, words[0], 0xF4292244, 6);
    if (!verify_same_bits(a, c, A13_C12_SAME_BITS)) {
      continue;
    }

    // d13
    d = II(d, a, b, c, words[7], 0x432AFF97, 10);
    if (!verify_different_bits(d, b, D13_B12_DIFFERENT_BITS)) {
      continue;
    }

    // c13
    c = II(c, d, a, b, words[14], 0xAB9423A7, 15);
    if (!verify_same_bits(c, a, C13_A13_SAME_BITS)) {
      continue;
    }

    // b13
    b = II(b, c, d, a, words[5], 0xFC93A039, 21);
    if (!verify_same_bits(b, d, B13_D13_SAME_BITS)) {
      continue;
    }

    // a14
    a = II(a, b, c, d, words[12], 0x655B59C3, 6);
    if (!verify_same_bits(a, c, A14_C13_SAME_BITS)) {
      continue;
    }

    // d14
    d = II(d, a, b, c, words[3], 0x8F0CCC92, 10);
    if (!verify_same_bits(d, b, D14_B13_SAME_BITS)) {
      continue;
    }

    // c14
    c = II(c, d, a, b, words[10], 0xFFEFF47D, 15);
    if (!verify_same_bits(c, a, C14_A14_SAME_BITS)) {
      continue;
    }

    // b14
    b = II(b, c, d, a, words[1], 0x85845DD1, 21);
    if (!verify_same_bits(b, d, B14_D14_SAME_BITS)) {
      continue;
    }

    // a15
    a = II(a, b, c, d, words[8], 0x6FA87E4F, 6);
    if (!verify_same_bits(a, c, A15_C14_SAME_BITS)) {
      continue;
    }

    // d15
    d = II(d, a, b, c, words[15], 0xFE2CE6E0, 10);
    if (!verify_same_bits(d, b, D15_B14_SAME_BITS)) {
      continue;
    }

    // c15
    c = II(c, d, a, b, words[6], 0xA3014314, 15);
    if (!verify_same_bits(c, a, C15_A15_SAME_BITS)) {
      continue;
    }

    // b15
    b = II(b, c, d, a, words[13], 0x4E0811A1, 21);
    if (!verify_different_bits(b, d, B15_D15_DIFFERENT_BITS)) {
      continue;
    }

    // a16
    a = II(a, b, c, d, words[4], 0xF7537E82, 6);
    if (!verify_one_bits(a, A16_ONE_BITS)) {
      continue;
    }
    if (!verify_same_bits(a, c, A16_C15_SAME_BITS)) {
      continue;
    }

    // d16
    d = II(d, a, b, c, words[11], 0xBD3AF235, 10);
    if (!verify_one_bits(d, D16_ONE_BITS)) {
      continue;
    }
    if (!verify_same_bits(d, b, D16_B15_SAME_BITS)) {
      continue;
    }

    for (size_t j = 0; j < 16; j++) {
      candidates[idx * 16 + j] = words[j];
    }

    found[idx] = true;

    break;
  }
}

/**
 * Host main routine
 */
extern "C" {

int generate_candidates_cuda(const uint32_t *state, uint32_t *candidates,
                             bool *found, const size_t iterations,
                             const size_t threadsPerBlock,
                             const size_t blockDim, const size_t seed) {
  // Error code to check return values for CUDA callsd
  hipError_t err = hipSuccess;

  size_t batch_size = threadsPerBlock * blockDim;

  uint32_t *d_state = NULL;
  err = hipMalloc((void **)&d_state, sizeof(uint32_t) * STATE_SIZE_WORDS);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector state (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  uint32_t *d_candidates = NULL;
  err = hipMalloc((void **)&d_candidates,
                   sizeof(uint32_t) * batch_size * CANDIDATES_SIZE_WORDS);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to allocate device vector candidates (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  bool *d_found = NULL;
  err = hipMalloc((void **)&d_found, sizeof(bool) * batch_size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector found (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_state, state, sizeof(uint32_t) * STATE_SIZE_WORDS,
                   hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector state from host to device (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  generate_candidates<<<blockDim, threadsPerBlock>>>(d_state, d_candidates,
                                                     d_found, iterations, seed);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to launch validateCandidates kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipDeviceSynchronize();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to synchronize the device! error=%s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(candidates, d_candidates, sizeof(uint32_t) * batch_size * 16,
                   hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector candidates from device to host (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(found, d_found, sizeof(uint8_t) * batch_size,
                   hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector found from device to host (error code "
            "%s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_candidates);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector candidates(error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_found);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector found (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_state);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector state (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipDeviceReset();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to deinitialize the device! error=%s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  return 0;
}
}
